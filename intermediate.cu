#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result){
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__
void init(float num, float* a, int N){
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;
    for (int i = index; i < N; i += stride){
        a[i] = num;
    }
}

__global__
void add(float* result, float* a, float* b, int N){
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    for (int i = index; i < N; i += stride){
        result[i] = a[i] + b[i];
    }
}

int main(){
    int deviceId;
    hipGetDevice(&deviceId);
    
    int numberOfSMs; // streaming multiprocessors
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    const int N = 2 << 24;
    size_t size = N*sizeof(float);

    float *device_a, *device_b, *device_c, *host_c;

    hipMalloc(&device_a, size);
    hipMalloc(&device_b, size);
    hipMalloc(&device_c, size);
    hipHostMalloc(&host_c, size);

    size_t threadsPerBlock = 256;
    size_t numberOfBlocks = 32*numberOfSMs;

    hipError_t addVectorsErr;
    hipError_t asyncErrInit;
    hipError_t asyncErrAdd;

    hipStream_t stream_a, stream_b, stream_c;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    
    initWith<<<numberOfBlocks, threadsPerBlock, 0, stream_a>>>(3, device_a, N);
    initWith<<<numberOfBlocks, threadsPerBlock, 0, stream_b>>>(4, device_b, N);
    initWith<<<numberOfBlocks, threadsPerBlock, 0, stream_c>>>(0, device_c, N);

    checkCuda(hipDeviceSynchronize());

    for (int i = 0; i < 4; i++){
        hipStream_t stream;
        hipStreamCreate(&stream);
        
        addVectorsInto<<<numberOfBlocks/4, threadsPerBlock, 0, stream>>>(&device_c[i*N/4], &device_a[i*N/4], &device_b[i*N/4], N/4);
        hipMemcpyAsync(&host_c[i*N/4], &device_c[i*N/4], size/4, hipMemcpyDeviceToHost, stream);
        hipStreamDestroy(stream);
    }

    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());
    
    hipStreamDestroy(stream_a);
    hipStreamDestroy(stream_b);
    hipStreamDestroy(stream_c);

    checkCuda(hipFree(device_a));
    checkCuda(hipFree(device_b));
    checkCuda(hipFree(device_c));
    checkCuda(hipHostFree(host_c));
}
